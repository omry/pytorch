#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <c10/macros/Macros.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/foreach/Utils.cuh>
#include <ATen/native/cuda/foreach/MultiTensorApply.cuh>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

namespace {

template<typename x_t, typename out_t>
struct AddScalarFunctor {
    __device__ void operator() (
        int chunk_size,
        TensorListMetadata<2>& tl,
        x_t scalar) {
            int tensor_loc = tl.block_to_tensor[blockIdx.x];
            int chunk_idx = tl.block_to_chunk[blockIdx.x];
            int n = tl.sizes[tensor_loc];

            x_t* x = (x_t*)tl.addresses[0][tensor_loc];
            x += chunk_idx * chunk_size;

            out_t* out = (out_t*)tl.addresses[1][tensor_loc];
            out += chunk_idx * chunk_size;

            n -= chunk_idx * chunk_size;

            x_t r_x[ILP];
            out_t r_out[ILP];

            // to make things simple, we put aligned case in a different code path
            if(n % ILP == 0 && chunk_size % ILP == 0 && is_aligned(x) && is_aligned(out)) {
                for(int i_start = threadIdx.x; i_start * ILP < n && i_start * ILP < chunk_size; i_start += blockDim.x) {
                    // load
                    load_store(r_x, x, 0 , i_start);
#pragma unroll
                    for(int ii = 0; ii < ILP; ii++) {
                        r_out[ii] = static_cast<x_t>(r_x[ii]) + scalar;
                    }
                    // store
                    load_store(out, r_out, i_start, 0);
                }
            }
            else {
                // Non-divergent exit condition for __syncthreads, not necessary here
                for(int i_start = 0; i_start < n && i_start < chunk_size; i_start += blockDim.x * ILP) {
#pragma unroll
                    for(int ii = 0; ii < ILP; ii++) {
                        r_x[ii] = 0;
                        int i = i_start + threadIdx.x + ii * blockDim.x;
                        if(i < n && i < chunk_size) {
                            r_x[ii] = x[i];
                        }
                    }
#pragma unroll
                    for(int ii = 0; ii < ILP; ii++) {
                        r_out[ii] = static_cast<x_t>(r_x[ii]) + scalar;
                    }
#pragma unroll
                    for(int ii = 0; ii < ILP; ii++) {
                        int i = i_start + threadIdx.x + ii * blockDim.x;
                        if(i < n && i < chunk_size)
                            out[i] = r_out[ii];
                    }
                }
            }
        }
};

} // namespace

std::vector<Tensor> foreach_tensor_add_scalar_kernel_cuda(TensorList tensors, Scalar scalar) {
    if (tensors.size() == 0) {
        return std::move(tensors.vec());
    }

    TORCH_CHECK(std::all_of(tensors.begin(), tensors.end(), [] (const Tensor& t) {
        return t.layout() == at::kStrided;
    }), "Only tensors with strided layouts are supported.");

    TORCH_CHECK(std::all_of(tensors.begin(), tensors.end(), [] (const Tensor& t) {
        return t.is_non_overlapping_and_dense();
    }), "Only non overlapping and dense tensors are supported.");

    std::vector<std::vector<at::Tensor>> tensor_lists; 
    std::vector<at::Tensor> vec_res;
    for (int i = 0; i < tensors.size(); i++) {
        vec_res.emplace_back(torch::empty_like(tensors[i]));
    }

    tensor_lists.emplace_back(std::move(tensors.vec()));
    tensor_lists.emplace_back(std::move(vec_res));

    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(kBool, kBFloat16, kHalf, tensors[0].scalar_type(), "foreach_tensor_add_scalar_kernel_cuda", [&]() {
        multi_tensor_apply<2>(tensor_lists, AddScalarFunctor<scalar_t, scalar_t>(), scalar.to<scalar_t>());
    });
    return tensor_lists[1];
}

}} // namespace at::native
